#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include "../common/common.h"
#include <hip/hip_runtime.h>





extern "C" 
__global__ void 
sumArraysOnGPU_1Dgrid1Dblock(float *A, 
                    float *B, float *C, const int N)
{

    // 1D general case: keeps working when arrays get big!
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    while (i < N){ 
        C[i] = A[i] + B[i];
        i += blockDim.x*gridDim.x;
    }
}


// grid 2D block 2D
__global__ void sumMatrixGPU_2Dgrid2Dblock(float *MatA, float *MatB, 
                                           float *MatC, 
                                                        int nx,int ny)
{

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}