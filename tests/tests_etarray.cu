#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <limits>

#include <cmath>


#include <stdio.h>
#include "../common/common.h"
#include <hip/hip_runtime.h>

#include "../src/etops1.hpp"
#include "../src/etscalar.hpp"
#include "../src/etmatrix.hpp"
#include "../src/etops2.hpp"

#include "../src/array_template.hpp"
#include "../src/arrayops.hpp"
#include "../src/tests_etarray.cuh"



using namespace std;


//const int THREADWORK = 2;

//const int THREADWORK = 1024;// << 5;
//const int THREADWORK = 1 << 12;

//const int maxThreads = 1 << 20;



// void checkResult(float *hostRef, float *gpuRef, const int N){
    
//     double epsilon = 1.0E-8;
//     bool match = 1;

//     for (int i = 0; i < N; i++){

//         if (abs(hostRef[i] - gpuRef[i]) > epsilon){
        
//             match = 0;
//             printf("Arrays do not match!\n");
//             printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
//                    gpuRef[i], i);
//             break;
//         }//else{
//           //  printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
//             //       gpuRef[i], i);
//         //}
//     }

//     if (match) printf("Arrays match.\n\n");

//     return;
// }


template <typename T>
void print (T const& c){

    int ncol = c.getncols();
    int nrow = c.getnrows();
    for (int i=0; i<nrow; ++i) {
        std::cout << '\n';
        for (int j=0; j<ncol; ++j) {
            std::cout << c(i,j) << ' ';
        }
    }
    std::cout << "\ndone \n" << std::endl;
}

template <typename T, typename R>
void compare (T const& c, R const& z, 
                const std::string& input){
    // this string pass makes a copy ;-)

    int ncols = c.getncols();
    int nrows = c.getnrows();
    std::cout << "checking simple " << input <<"\n";
    for (size_t i = 0; i < nrows; i++) {
        for (size_t j = 0; j < ncols; j++) {
            assert( c(i,j) == z(i,j) );
        }
    }
    std::cout << "simple " <<  input <<" OK \n\n";
}




// __global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){

//     // 1D general case: keeps working when arrays get big:
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     while (i < N){ 
//         C[i] = A[i] + B[i];
//         i += blockDim.x*gridDim.x;
//     }
// }


__global__ void helloFromGPUthread(){

    int i = threadIdx.x;
    printf("Hellow World GPU id %d \n", i);
}

void helloFromCpu(){

    printf("---------------------\n");
    for (int i =0; i<1; i++)
    {
        printf("Hello World from CPU!\n");
    }
    printf("---------------------\n");
}



void devprops(int dev, 
                int driverVersion, 
                int runtimeVersion,
                hipDeviceProp_t deviceProp){
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);
    printf("  Total amount of global memory:                 %.2f GBytes (%llu "
           "bytes)\n", (float)deviceProp.totalGlobalMem / pow(1024.0, 3),
           (unsigned long long)deviceProp.totalGlobalMem);
    printf("  GPU Clock rate:                                %.0f MHz (%0.2f "
           "GHz)\n", deviceProp.clockRate * 1e-3f,
           deviceProp.clockRate * 1e-6f);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
           deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize)
    {
        printf("  L2 Cache Size:                                 %d bytes\n",
               deviceProp.l2CacheSize);
    }

    printf("  Max Texture Dimension Size (x,y,z)             1D=(%d), "
           "2D=(%d,%d), 3D=(%d,%d,%d)\n", deviceProp.maxTexture1D,
           deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
           deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
           deviceProp.maxTexture3D[2]);
    printf("  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, "
           "2D=(%d,%d) x %d\n", deviceProp.maxTexture1DLayered[0],
           deviceProp.maxTexture1DLayered[1], deviceProp.maxTexture2DLayered[0],
           deviceProp.maxTexture2DLayered[1],
           deviceProp.maxTexture2DLayered[2]);
    printf("  Total amount of constant memory:               %lu bytes\n",
           deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %lu bytes\n",
           deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n",
           deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
           deviceProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
           deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
           deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %lu bytes\n",
           deviceProp.memPitch);
    exit(EXIT_SUCCESS);
}



int etarraybasic (){


    /* set up device: 
       nvcc required! */
//     int dev = 0, driverVersion = 0, runtimeVersion = 0;
//     hipDeviceProp_t deviceProp;
//     CHECK(hipGetDeviceProperties(&deviceProp, dev));
//     printf("Using Device %d: %s\n", dev, deviceProp.name);
//     CHECK(hipSetDevice(dev));



    int np = 5;
    //int nrows = np;
    //int ncols = np;
    Array<double> a(np,np), b(np,np), c(np,np), d(np,np);

    Array2D<double> x(np,np), y(np,np), z(np,np), w(np,np);

    // initialize arrays with some values
    for (int i=0; i<np; ++i) {
         for (int j=0; j<np; ++j) {
            a(i,j) = static_cast <double>(i*(j+1));
            a(i,j) = 1.;//a(i,j) + 5.;
            b(i,j) = a(i,j)+a(i,j);
            c(i,j) = a(i,j)+b(i,j);

            x(i,j) = static_cast <double>(i*(j+1));
            x(i,j) = 1.;//x(i,j) + 5.;
            y(i,j) = x(i,j)+x(i,j);
            z(i,j) = x(i,j)+y(i,j);
         }
    }
    //...
    
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\n Welcome to the Test Folder! \n";
    std::cout << "\n\nTesting basic array class  \n";
    std::cout << "against  \n";
    std::cout << "Expression Template array class  \n";
    std::cout << "-----------------------------\n\n";
    std::cout << "\n\n-----------------------------\n";

    std::cout << "b: ";
    print(b);

    std::cout << "c: ";
    print(c);


    std::cout << "y: ";
    print(y);

    std::cout << "z: ";
    print(z);



    z = matmul(x,y);
    std::cout << "xfter z=matmul(x,y) ";
    print(z);
    d = 1.;

    c = matmul(a,b);
    std::cout << "after c=matmul(a,b) ";
    print(c);
    d = 1.;

    compare(c,z,"matmul");

    std::cout << "c = b + a: \n";
    std::cout.flush();
    c = b + a;
    std::cout << "c = b + a done \n";
    std::cout.flush();
    print(c);

    std::cout << "z = y + x: \n";
    std::cout.flush();
    z = y + x;
    std::cout << "z = y + x done \n";
    std::cout.flush();
    print(z);


    compare(c,z,"elementwise addition");

    
    a = 1.2 * a;  //expression template supports scalars
    std::cout << "1.2*a ";
    print(a);

    x = 1.2 * x;  //expression template supports scalars
    std::cout << "1.2*x ";
    print(x);


    compare(a,x,"scalar + matrix addition");

    std::cout << "a = c * a";
    a = c * a;
    print(a);
    std::cout << "a = c * a: ";
    print(a);


    std::cout << "x = z * x";
    x = z * x;
    print(x);
    std::cout << "x = z * x: ";
    print(x);

    compare(a,x,"elementwise matrix multiplication");


    a = 1.2*a + a*b;
    std::cout << "1.2*a + a*b: ";
    print(a);


    x = 1.2*x + x*y;
    std::cout << "1.2*x + x*y: ";
    print(x);

    compare(a,x,"scalar * matrix multiplication");

    a = b;
    std::cout << "after a = b: ";
    print(b);
    print(a);

    x = y;
    std::cout << "xfter x = y: ";
    print(y);
    print(x);

    compare(a,y,"crossed eualtiy");
    compare(b,x,"crossed equality");

    c = (a*b)+c;
    std::cout << "after c= (a*b)+c ";
    print(c);


    z = (x*y)+z;
    std::cout << "xfter z= (x*y)+z ";
    print(z);
    compare(c,z,"(mat * mat) + mat combined");


    c = (a+b)+c;
    std::cout << "after c= (a+b)+c ";
    print(c);
    
    z = (x+y)+z;
    std::cout << "xfter z= (x+y)+z ";
    print(z);

    compare(c,z,"(mat + mat) + mat combined");


    d=1.;
    c = matmul(d,c);
    std::cout << "after c=matmul(d,c) ";
    print(c);

    w=1.;
    z = matmul(w,z);
    std::cout << "xfter z=matmul(w,z) ";
    print(z);
    
    compare(c,z,"(matmul");
    



    std::cout << "b: ";
    print(b);
    //d = .5*b;
    compare(b,y,"b,y");

    //c = matmul(d,c);
    c = matmul((c+.5*d),d); //still no support for combined & cached terms
    std::cout << "after c=matmul((1.+b),c) ";
    print(c);

    
    std::cout << "y: ";
    print(y);
    //w = .5*y;
    compare(d,w,"d,w");

    //z = matmul(w,z);
    z = matmul((z+.5*w),w); //still no support for zomyinew & zxzhew terms
    std::cout << "xfter z=matmul((1.+y),z) ";
    print(z);


    compare(c,z,"compound mat + (scalar * mat )matmul mat");



    b=1;
    
    c=1.;
    c = matmul(b,c);
    std::cout << "after b=1.;c=1; c=matmul(b,c); c:";
    print(c);


    y=1;
    
    z=1.;
    z = matmul(y,z);
    std::cout << "xfter y=1.;z=1; z=matmul(y,z); z:";
    print(z);


    compare(c,z,"matmul");


    
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\nTesting basic array class  \n";
    std::cout << "against  \n";
    std::cout << "Expression Template array class  \n";

    std::cout << "result:  \n";
    std::cout << "         all tests pass  \n";
    std::cout << "-----------------------------\n\n";


    // printf("---------------------\n");
    // // GPU call:
    // helloFromGPU<<<1, 10>>>();
    // //CHECK(hipDeviceSynchronize());
    // helloFromGPUthread<<<1, 10>>>();
    // CHECK(hipDeviceReset());
    // printf("---------------------\n");



    helloFromCpu();
    helloFromGPUthread<<<1, 10>>>();
    CHECK(hipDeviceReset());

    return 0;

}