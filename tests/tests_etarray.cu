#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <limits>

#include <cmath>


#include <stdio.h>
#include "../common/common.h"
#include <hip/hip_runtime.h>

#include "../src/etops1.hpp"
#include "../src/etscalar.hpp"
#include "../src/etmatrix.hpp"
#include "../src/etops2.hpp"

#include "../src/array_template.hpp"
#include "../src/arrayops.hpp"
#include "../src/tests_etarray.cuh"



using namespace std;
using std::cout;
using std::endl;


//const int THREADWORK = 2;

//const int THREADWORK = 1024;// << 5;
//const int THREADWORK = 1 << 12;

//const int maxThreads = 1 << 20;



// void checkResult(float *hostRef, float *gpuRef, const int N){
    
//     double epsilon = 1.0E-8;
//     bool match = 1;

//     for (int i = 0; i < N; i++){

//         if (abs(hostRef[i] - gpuRef[i]) > epsilon){
        
//             match = 0;
//             printf("Arrays do not match!\n");
//             printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
//                    gpuRef[i], i);
//             break;
//         }//else{
//           //  printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
//             //       gpuRef[i], i);
//         //}
//     }

//     if (match) printf("Arrays match.\n\n");

//     return;
// }


template <typename T>
void print (T const& c){

    int ncol = c.getncols();
    int nrow = c.getnrows();
    for (int i=0; i<nrow; ++i) {
        std::cout << '\n';
        for (int j=0; j<ncol; ++j) {
            std::cout << c(i,j) << ' ';
        }
    }
    printf("\ndone \n");
    //std::cout<< "\ndone \n" <<std::endl;
}

template <typename T, typename R>
void compare (T const& c, R const& z, 
                const std::string& input){
    // this string pass makes a copy ;-)

    int ncols = c.getncols();
    int nrows = c.getnrows();
    printf("checking simple  %s  \n",input.c_str());
    //std::cout << "checking simple " << input <<"\n";
    for (size_t i = 0; i < nrows; i++) {
        for (size_t j = 0; j < ncols; j++) {
            assert( c(i,j) == z(i,j) );
        }
    }
    printf("simple %s OK \n\n",input.c_str());
    //std::cout << "simple " <<  input <<" OK \n\n";
}




// __global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){

//     // 1D general case: keeps working when arrays get big:
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     while (i < N){ 
//         C[i] = A[i] + B[i];
//         i += blockDim.x*gridDim.x;
//     }
// }


__global__ void helloFromGPUthread(){

    int i = threadIdx.x;
    printf("Hellow World from GPU id %d !\n", i);
    printf("threadIdx = %d \n",threadIdx.x);
    printf("blockIdx = %d \n",blockIdx.x);
    printf("blockDim = %d \n",blockDim.x);
}

void helloFromCpu(){

    printf("---------------------\n");
    for (int i =0; i<1; i++)
    {
        printf("Hello World from CPU!\n");
    }
    printf("---------------------\n");
}




//int etarraybasic (SolverManager& SM){
int etarraybasic (){



    int np = 10;

    //expression template matrices:
    Array<float> a(np,np), b(np,np), c(np,np), d(np,np);


    //compare with traditional overloaded matrices:
    Array2D<float> x(np,np), y(np,np), z(np,np), w(np,np);

    // initialize arrays with some values
    for (int i=0; i<np; ++i) {
         for (int j=0; j<np; ++j) {
            a(i,j) = static_cast <float>(i*(j+1));
            a(i,j) = 1.;//a(i,j) + 5.;
            b(i,j) = a(i,j)+a(i,j);
            c(i,j) = a(i,j)+b(i,j);

            x(i,j) = static_cast <float>(i*(j+1));
            x(i,j) = 1.;//x(i,j) + 5.;
            y(i,j) = x(i,j)+x(i,j);
            z(i,j) = x(i,j)+y(i,j);
         }
    }
    //...
    
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\n Welcome to the Test Folder! \n";
    std::cout << "\n\nTesting basic array class  \n";
    std::cout << "against  \n";
    std::cout << "Expression Template array class  \n";
    std::cout << "-----------------------------\n\n";
    std::cout << "\n\n-----------------------------\n";

    std::cout << "b: ";
    print(b);

    std::cout << "c: ";
    print(c);


    std::cout << "y: ";
    print(y);

    std::cout << "z: ";
    print(z);



    z = matmul(x,y);
    std::cout << "xfter z=matmul(x,y) ";
    print(z);
    d = 1.;

    c = matmul(a,b);
    std::cout << "after c=matmul(a,b) ";
    print(c);
    d = 1.;

    compare(c,z,"matmul");

    std::cout << "c = b + a: \n";
    std::cout.flush();
    c = b + a;
    std::cout << "c = b + a done \n";
    std::cout.flush();
    print(c);

    std::cout << "z = y + x: \n";
    std::cout.flush();
    z = y + x;
    std::cout << "z = y + x done \n";
    std::cout.flush();
    print(z);


    compare(c,z,"elementwise addition");

    
    a = 1.2f * a;  //expression template supports scalars
    std::cout << "1.2*a ";
    print(a);

    x = 1.2f * x;  //expression template supports scalars
    std::cout << "1.2*x ";
    print(x);


    compare(a,x,"scalar + matrix addition");

    std::cout << "a = c * a";
    a = c * a;
    print(a);
    std::cout << "a = c * a: ";
    print(a);


    std::cout << "x = z * x";
    x = z * x;
    print(x);
    std::cout << "x = z * x: ";
    print(x);

    compare(a,x,"elementwise matrix multiplication");


    a = 1.2f*a + a*b;
    std::cout << "1.2*a + a*b: ";
    print(a);


    x = 1.2f*x + x*y;
    std::cout << "1.2*x + x*y: ";
    print(x);

    compare(a,x,"scalar * matrix multiplication");

    a = b;
    std::cout << "after a = b: ";
    print(b);
    print(a);

    x = y;
    std::cout << "xfter x = y: ";
    print(y);
    print(x);

    compare(a,y,"crossed eualtiy");
    compare(b,x,"crossed equality");

    c = (a*b)+c;
    std::cout << "after c= (a*b)+c ";
    print(c);


    z = (x*y)+z;
    std::cout << "xfter z= (x*y)+z ";
    print(z);
    compare(c,z,"(mat * mat) + mat combined");


    c = (a+b)+c;
    std::cout << "after c= (a+b)+c ";
    print(c);
    
    z = (x+y)+z;
    std::cout << "xfter z= (x+y)+z ";
    print(z);

    compare(c,z,"(mat + mat) + mat combined");


    d=1.f;
    c = matmul(d,c);
    std::cout << "after c=matmul(d,c) ";
    print(c);

    w=1.f;
    z = matmul(w,z);
    std::cout << "xfter z=matmul(w,z) ";
    print(z);
    
    compare(c,z,"(matmul");
    



    std::cout << "b: ";
    print(b);
    //d = .5*b;
    compare(b,y,"b,y");

    //c = matmul(d,c);
    c = matmul((c+.5f*d),d); //combined & cached terms are supported
    std::cout << "after c=matmul((1.+b),c) ";
    print(c);

    
    std::cout << "y: ";
    print(y);
    //w = .5*y;
    compare(d,w,"d,w");

    //z = matmul(w,z);
    z = matmul((z+.5f*w),w); //compare with traditional overload
    std::cout << "xfter z=matmul((1.+y),z) ";
    print(z);


    compare(c,z,"compound mat + (scalar * mat )matmul mat");



    b=1;
    
    c=1.;
    c = matmul(b,c);
    std::cout << "after b=1.;c=1; c=matmul(b,c); c:";
    print(c);


    y=1;
    
    z=1.;
    z = matmul(y,z);
    std::cout << "xfter y=1.;z=1; z=matmul(y,z); z:";
    print(z);


    compare(c,z,"matmul");


    
    std::cout << "\n\n-----------------------------\n";
    std::cout << "\n\nTesting basic array class  \n";
    std::cout << "against  \n";
    std::cout << "Expression Template array class  \n";

    std::cout << "result:  \n";
    std::cout << "         all tests pass  \n";
    std::cout << "-----------------------------\n\n";


    // printf("---------------------\n");
    // // GPU call:
    // helloFromGPU<<<1, 10>>>();
    // //CHECK(hipDeviceSynchronize());
    // helloFromGPUthread<<<1, 10>>>();
    // CHECK(hipDeviceReset());
    // printf("---------------------\n");



    helloFromCpu();
    helloFromGPUthread<<<1, 10>>>();

    CHECK(hipDeviceReset());

    //exit(EXIT_SUCCESS);
    return 0;

}